#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "hipblas.h"
#include ""
#include <iostream>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <ctime>
#include <math.h>
#include <queue>
#include <string.h>
#include <stdlib.h>
#include <vector>

using namespace std;

struct Point {
	float		dimensions[128];
	int			cluster;
	int			noise;  //-1 noise;
    string      img;
};

float __device__ dev_euclidean_distance(const Point &src, const Point &dest) {
    float res = 0.0;
    for(int i=0; i<128; i++){
        res += (src.dimensions[i] - dest.dimensions[i]) * (src.dimensions[i] - dest.dimensions[i]);
    }
	return sqrt(res);
}

/*to get the total list*/
void __global__ dev_region_query(Point* sample, int num, int* neighbors, float eps, int min_nb) {

	unsigned int	tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int	line,col,pointer = tid;
	unsigned int	count;

	while (pointer < num * num) {//全场唯一id
		line = pointer / num;
		col = pointer % num;
		float radius;
		if (line <= col) {
			radius = dev_euclidean_distance(sample[line], sample[col]);
			if (radius <= eps) {
				neighbors[pointer] = 1;
			}
			neighbors[col * num + line] = neighbors[pointer];//对角线
		}
		pointer += blockDim.x * gridDim.x;
	}
	__syncthreads();

	pointer = tid;
	while (pointer < num) {
		count = 1;
		line = pointer * num;
		for (int i = 0; i < num; i++) {
			if (pointer != i && neighbors[line+i]) {//包含p点邻域元素个数
				count++;
			}
		}
		if (count >= min_nb) {
			sample[pointer].noise++;
		}
		pointer += blockDim.x * gridDim.x;
	}
}

void host_algorithm_dbscan(Point* host_sample, int num, float eps, int min_nb, int block_num, int thread_num) {
	/*sample*/
	Point* cuda_sample;
	hipMalloc((void**)&cuda_sample, num * sizeof(Point));
	hipMemcpy(cuda_sample, host_sample, num * sizeof(Point), hipMemcpyHostToDevice);

	/*neighbor list*/
	int *host_neighbor = new int[num*num]();
	int *dev_neighbor;
	hipMalloc((void**)&dev_neighbor, num * num * sizeof(int));

	dev_region_query << <block_num, thread_num >> > (cuda_sample, num, dev_neighbor, eps, min_nb);

	hipMemcpy(host_sample, cuda_sample, num * sizeof(Point), hipMemcpyDeviceToHost);
	hipMemcpy(host_neighbor, dev_neighbor, num * num * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(cuda_sample);hipFree(dev_neighbor);

	queue<int> expand;
	int cur_cluster = 0;

	for (int i = 0; i < num; i++) {
		if (host_sample[i].noise >= 0 && host_sample[i].cluster < 1) {
			host_sample[i].cluster = ++cur_cluster;
			int src = i * num;
			for (int j = 0; j < num; j++) {
				if (host_neighbor[src + j]) {
					host_sample[j].cluster = cur_cluster;
					expand.push(j);
				}
			}

			while (!expand.empty()) {/*expand the cluster*/
				if (host_sample[expand.front()].noise >= 0) {
					src = expand.front() * num;
					for (int j = 0; j < num; j++) {
						if (host_neighbor[src + j] && host_sample[j].cluster < 1) {
							host_sample[j].cluster = cur_cluster;
							expand.push(j);
						}
					}
				}
				expand.pop();
			}
		}
	}

	ofstream fout;
    fout.open("result.html");
    for (int i = 0; i < num; i++) {
        fout <<"<img src='"<< host_sample[i].img << "'/>" <<host_sample[i].cluster<< endl;
    }
    fout.close();
}

// 读取文件行数
int countLines(const char *filename){
    ifstream fin(filename, ios::in);
    int n=0;
    string lineStr;
    while(getline(fin, lineStr)) n++;
    return n;
}

extern "C"
int main(int argc, char* argv[]) {
    const char* file_name = "data.csv";
    // 获取文件的行数
    int point_count = 100;

    Point *host_sample ;
    try{
        host_sample = new Point[point_count];
    }catch(std::bad_alloc){
        cout << "new array error" << endl;
    }
    // 然后将每行的数据读到Point结构体中
	int sample_num = 0;
	string lineStr;
	ifstream fin(file_name, ios::in);
	while(getline(fin, lineStr)){
	    stringstream ss(lineStr);
        vector<string> lineArray;
        string str;
        // 按照逗号分隔
        while (getline(ss, str, ','))
            lineArray.push_back(str);/* 将文件中每一行存入到vector中，其中lineArray[0]存放的是特征值 */
        // 分离出特征值即lineArray[0]后，是一个以“_”分割的字符串，解析出来存到Point结构体的dimensions中
        char *datas;
        const int len = lineArray[0].length();
        datas = new char[len + 1];
        strcpy(datas, lineArray[0].c_str());
	    const char dims[2] = "_";
        char *token;
        // 获取第一个子字符串
        token = strtok(datas, dims);
        // 继续获取其他的子字符串
        int i=0;
        while( token != NULL )
        {
            host_sample[sample_num].dimensions[i++] = atof(token);
            token = strtok(NULL, dims);
        }
        host_sample[sample_num].noise = -1;
        host_sample[sample_num].cluster = -1;
        host_sample[sample_num].img = lineArray[1];
        sample_num++;
        if(sample_num == point_count){
            break;
        }

	}
	cout << "------>TOTAL SAMPLE NUMB0->" << sample_num << "<-----" << endl;

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);

	host_algorithm_dbscan(host_sample, point_count, 0.87f, 2, 20, 100);
    delete []host_sample;

	hipEventRecord(end, 0);
	hipEventSynchronize(end);

	float time;
	hipEventElapsedTime(&time, start, end);
	cout<< file_name << " speed time: "<< time <<"ms\n"<<endl;

}