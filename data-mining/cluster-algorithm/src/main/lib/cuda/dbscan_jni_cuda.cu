#include "hip/hip_runtime.h"
#include "intellif_minning_dbscan_impl_DBSCANImpl.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "hipblas.h"
#include ""
#include <iostream>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <ctime>
#include <math.h>
#include <queue>
#include <string.h>
#include <stdlib.h>
#include <vector>
#include<stdio.h>
#include<algorithm>
#include<memory>

//API调用错误处理，可以接受CUDA的API函数调用作为参数
#define CHECK_ERROR(error) checkCudaError(error, __FILE__, __LINE__)
//检查CUDA Runtime状态码，可以接受一个指定的提示信息
#define CHECK_STATE(msg) checkCudaState(msg, __FILE__, __LINE__)

inline void checkCudaError(hipError_t error, const char *file, const int line)
{
   if (error != hipSuccess) {
      std::cerr << "CUDA CALL FAILED:" << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
}

inline void checkCudaState(const char *msg, const char *file, const int line)
{
   hipError_t error = hipGetLastError();
   if (error != hipSuccess) {
      std::cerr << "---" << msg << " Error---" << std::endl;
      std::cerr << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
}

using namespace std;

struct Point {
	float		dimensions[128];
	int			cluster;
	int			noise;  //-1 noise;
    //string      img;
};

float eps;//neighborhood radius
int min_nb;
int n;
Point *host_sample;
int block_num = 96;
int thread_num = 32;

float __device__ dev_euclidean_distance(const Point &src, const Point &dest) {
    float res = 0.0;
    for(int i=0; i<128; i++){
        res += (src.dimensions[i] - dest.dimensions[i]) * (src.dimensions[i] - dest.dimensions[i]);
    }
	return sqrt(res);
}

/*to get the total list*/
void __global__ dev_region_query(Point* sample, int num, int* neighbors, float eps, int min_nb) {

	unsigned int	tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int	line,col,pointer = tid;
	unsigned int	count;

	while (pointer < num * num) {//全场唯一id
		line = pointer / num;
		col = pointer % num;
		float radius;
		if (line <= col) {
			radius = dev_euclidean_distance(sample[line], sample[col]);
			if (radius <= eps) {
				neighbors[pointer] = 1;
			}
			neighbors[col * num + line] = neighbors[pointer];//对角线
		}
		pointer += blockDim.x * gridDim.x;
	}
	__syncthreads();

	pointer = tid;
	while (pointer < num) {
		count = 1;
		line = pointer * num;
		for (int i = 0; i < num; i++) {
			if (pointer != i && neighbors[line+i]) {//包含p点邻域元素个数
				count++;
			}
		}
		if (count >= min_nb) {
			sample[pointer].noise++;
		}
		pointer += blockDim.x * gridDim.x;
	}
}

void host_algorithm_dbscan() {
	int num = n;
	/*sample*/
	Point* cuda_sample;
	CHECK_ERROR(hipMalloc((void**)&cuda_sample, num * sizeof(Point)));
	CHECK_ERROR(hipMemcpy(cuda_sample, host_sample, num * sizeof(Point), hipMemcpyHostToDevice));

	/*neighbor list*/
	int *host_neighbor = new int[num*num]();
	int *dev_neighbor;
	CHECK_ERROR(hipMalloc((void**)&dev_neighbor, num * num * sizeof(int)));

	dev_region_query << <block_num, thread_num >> > (cuda_sample, num, dev_neighbor, eps, min_nb);
    hipDeviceSynchronize();
    CHECK_STATE("kernel call");

	CHECK_ERROR(hipMemcpy(host_sample, cuda_sample, num * sizeof(Point), hipMemcpyDeviceToHost));
	CHECK_ERROR(hipMemcpy(host_neighbor, dev_neighbor, num * num * sizeof(int), hipMemcpyDeviceToHost));

    hipFree(cuda_sample);
    hipFree(dev_neighbor);

	queue<int> expand;
	int cur_cluster = 0;

	for (int i = 0; i < num; i++) {
		if (host_sample[i].noise >= 0 && host_sample[i].cluster < 1) {
			host_sample[i].cluster = ++cur_cluster;
			int src = i * num;
			for (int j = 0; j < num; j++) {
				if (host_neighbor[src + j]) {
					host_sample[j].cluster = cur_cluster;
					expand.push(j);
				}
			}

			while (!expand.empty()) {/*expand the cluster*/
				if (host_sample[expand.front()].noise >= 0) {
					src = expand.front() * num;
					for (int j = 0; j < num; j++) {
						if (host_neighbor[src + j] && host_sample[j].cluster < 1) {
							host_sample[j].cluster = cur_cluster;
							expand.push(j);
						}
					}
				}
				expand.pop();
			}
		}
	}
}

// 读取文件行数
int countLines(const char *filename){
    ifstream fin(filename, ios::in);
    int n=0;
    string lineStr;
    while(getline(fin, lineStr)) n++;
    return n;
}

//删除字符串中空格，制表符tab等无效字符
string Trim(string& str)
{
	//str.find_first_not_of(" \t\r\n"),在字符串str中从索引0开始，返回首次不匹配"\t\r\n"的位置
	str.erase(0, str.find_first_not_of(" \t\r\n"));
	str.erase(str.find_last_not_of(" \t\r\n") + 1);
	return str;
}

JNIEXPORT jboolean JNICALL Java_intellif_minning_dbscan_impl_DBSCANImpl_initDatasFromFile
(JNIEnv *env, jobject obj, jstring jfile, jint jcount) {
	const char *file = env->GetStringUTFChars(jfile, NULL);// 从java 得到file
	n = (int)jcount;// 从java 得到 count

	try
	{
		host_sample = new Point[n];// 分配数组空间
	}
	catch (const std::exception& e)
	{
		cerr << "alloca arrays exception: " << e.what() << endl;
		exit(EXIT_FAILURE);
	}

	ifstream fin(file); //打开文件流操作
	if (!fin)
	{
		cout << "file not found" << endl;
		exit(EXIT_FAILURE);
	}
	string line;
	int point_count = 0;
	while (getline(fin, line))   //整行读取，换行符“\n”区分，遇到文件尾标志eof终止读取
	{
		istringstream sin(line); //将整行字符串line读入到字符串流istringstream中
		vector<string> fields; //声明一个字符串向量
		string field;
		while (getline(sin, field, ',')) //将字符串流sin中的字符读入到field字符串中，以逗号为分隔符
		{
			fields.push_back(field); //将刚刚读取的字符串添加到向量fields中
		}
		string alls = Trim(fields[0]); // 文件中每行都是一个字符串

        //清除掉向量fields中第一个元素的无效字符，并赋值给变量features(特征值字符串)
        size_t pos = alls.find(" ");
        string features = alls.substr(pos + 1);// 特征值转换，并初始化temp，此时的temp顺序是按照文件中读取的顺序
        istringstream featurestream(features);
        string feature;
        int dims = 0;
        while (getline(featurestream, feature, ' '))
        {
            host_sample[point_count].dimensions[dims++] = stof(feature);
        }
		//host_sample[point_count].img = Trim(fields[1]); //清除掉向量fields中第二个元素的无效字符，并赋值给变量img
		host_sample[point_count].noise = -1;
        host_sample[point_count].cluster = -1;
		point_count++;
		if (point_count >= n)
		{
			break;
		}
	}

	env->ReleaseStringUTFChars(jfile, file);

	cout << "init points from file success" << endl;

	return (jboolean)true;
}

// dbscan
JNIEXPORT void JNICALL Java_intellif_minning_dbscan_impl_DBSCANImpl_runDBSCAN
(JNIEnv *env, jobject obj, jfloat jeps, jint jminPts) {
	eps = (float)jeps;
	min_nb = (int)jminPts;

	clock_t start, finish;
	start = clock();

	// 聚类
	host_algorithm_dbscan();

	finish = clock();

	cout << "dbscan success" << endl;

	cout << n << " speed time: " << (finish - start)*1.0 / CLOCKS_PER_SEC << "s\n" << endl;
}

JNIEXPORT jstring JNICALL Java_intellif_minning_dbscan_impl_DBSCANImpl_saveDBSCAN
(JNIEnv *env, jobject obj) {

	ofstream fout;
    char resultFile[128];
    sprintf(resultFile, "%d_result.csv", n);
    fout.open(resultFile);
    for (int i = 0; i < n; i++) {
        fout << i << "," << host_sample[i].cluster << endl;
    }
    fout.close();

	// 释放内存
	delete []host_sample;

	cout << "save result success" << endl;

	return env->NewStringUTF(resultFile);
}
