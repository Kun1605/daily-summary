
#include <hip/hip_runtime.h>
extern "C"
__global__ void cudaGetNeighbors(float* xs, float* ys, int* vis, int len, int* neighbors, double minEps, int minPts) {

	unsigned int	tid	= blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int	src;
	unsigned int	dest;
	unsigned int	point_id = tid;
	unsigned int	neighborscnt;

	while (point_id < len * len) {
		src = point_id / len;
		dest = point_id % len;
		float dist;
		if (src <= dest) {
			float srcX = xs[src];
			float destX = xs[dest];
			float srcY = ys[src];
			float destY = ys[dest];
			float xRes = srcX - destX;
			float yRes = srcY - destY;
			dist = xRes * xRes + yRes * yRes;
			if (dist <= minEps * minEps) {
				neighbors[point_id] = 1;
			}
			neighbors[dest * len + src] = neighbors[point_id];
		}
		point_id += blockDim.x * gridDim.x;
	}

	__syncthreads();

	point_id = tid;
	while (point_id < len) {
		neighborscnt = 1;
		src = point_id * len;
		for (int i = 0; i < len; i++) {
			if (point_id != i) {
				if (neighbors[src + i]) {
					neighborscnt++;
				}
			}
		}
		if (neighborscnt >= minPts) {
			vis[point_id]++;
		}
		point_id += blockDim.x * gridDim.x;
	}
}
