#include "hip/hip_runtime.h"
#include "CudaDBSCAN.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "hipblas.h"
#include ""
#include <iostream>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <ctime>
#include <math.h>
#include <queue>
#include <string.h>
#include <stdlib.h>

using namespace std;

struct Point {
	float		dimensions[128];
	int			cluster; // cluster id
	int			noise;  // -1 noise, 0 core
    string      img;
    int         id;
};

/* to get distance between two points */
float __device__ dev_euclidean_distance(const Point &src, const Point &dest) {
    float res = 0.0;
    for(int i=0; i<128; i++){
        res += (src.dimensions[i] - dest.dimensions[i]) * (src.dimensions[i] - dest.dimensions[i]);
    }
	return sqrt(res);
}

/* the gpu function */
void __global__ dev_region_query(Point* sample, int num, int* neighbors, float eps, int min_nb) {

	unsigned int	tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int	line,col,pointer = tid;
	unsigned int	count;

	while (pointer < num * num) {//全场唯一id
		line = pointer / num;
		col = pointer % num;
		float radius;
		if (line <= col) {
			radius = dev_euclidean_distance(sample[line], sample[col]);
			if (radius <= eps) {
				neighbors[pointer] = 1;
			}
			neighbors[col * num + line] = neighbors[pointer];//对角线
		}
		pointer += blockDim.x * gridDim.x;
	}
	__syncthreads();

	pointer = tid;
	while (pointer < num) {
		count = 1;
		line = pointer * num;
		for (int i = 0; i < num; i++) {
			if (pointer != i && neighbors[line+i]) {//包含p点邻域元素个数
					count++;
			}
		}
		if (count >= min_nb) {
			sample[pointer].noise++;
		}
		pointer += blockDim.x * gridDim.x;
	}
}

void host_algorithm_dbscan(Point* host_sample, int num, float eps, int min_nb) {
	/*sample*/
	Point* cuda_sample;
	hipMalloc((void**)&cuda_sample, num * sizeof(Point));
	hipMemcpy(cuda_sample, host_sample, num * sizeof(Point), hipMemcpyHostToDevice);

	/*neighbor list*/
	int *host_neighbor = new int[num*num]();
	int *dev_neighbor;
	hipMalloc((void**)&dev_neighbor, num * num * sizeof(int));

    /* run on gpu */
    int block_num = 10;
	int thread_num = 100;
	dev_region_query << <block_num, thread_num >> > (cuda_sample, num, dev_neighbor, eps, min_nb);

	hipMemcpy(host_sample, cuda_sample, num * sizeof(Point), hipMemcpyDeviceToHost);
	hipMemcpy(host_neighbor, dev_neighbor, num * num * sizeof(int), hipMemcpyDeviceToHost);

	queue<int> expand;
	int cur_cluster = 0;

	for (int i = 0; i < num; i++) {
		if (host_sample[i].noise >= 0 && host_sample[i].cluster < 1) {
			host_sample[i].cluster = ++cur_cluster;
			int src = i * num;
			for (int j = 0; j < num; j++) {
				if (host_neighbor[src + j]) {
					host_sample[j].cluster = cur_cluster;
					expand.push(j);
				}
			}

			while (!expand.empty()) {/*expand the cluster*/
				if (host_sample[expand.front()].noise >= 0) {
					src = expand.front() * num;
					for (int j = 0; j < num; j++) {
						if (host_neighbor[src + j] && host_sample[j].cluster < 1) {
							host_sample[j].cluster = cur_cluster;
							expand.push(j);
						}
					}
				}
				expand.pop();
			}
		}
	}
	hipFree(cuda_sample);hipFree(dev_neighbor);

	ofstream fout;
    fout.open("result.html");
    for (int i = 0; i < num; i++) {
    	fout << host_sample[i].id << " " <<host_sample[i].cluster<< endl;
    }
    fout.close();
}

extern "C"
JNIEXPORT void JNICALL Java_CudaDBSCAN_runDBSCAN
  (JNIEnv *env, jobject obj, jobject objectList, jfloat eps, jint minPts)
  {
        const char *str ="enter native method\n";
        cout << str <<endl;

        /* get the list class */
        jclass cls_list = env->GetObjectClass(objectList);
        if(cls_list == NULL){
            cout << "not find class\n" << endl;
        }

        /* method in class List  */
        jmethodID list_get = env->GetMethodID(cls_list, "get", "(I)Ljava/lang/Object;");
        jmethodID list_size = env->GetMethodID(cls_list, "size", "()I");
        if(list_get == NULL){
            cout << "not find get method\n" << endl;
        }
        if(list_size == NULL){
            cout << "not find size method\n" << endl;
        }

        /* jni invoke list.get to get points count */
        int len = static_cast<int>(env->CallIntMethod(objectList, list_size));
        if(len > 0){
            cout << len << endl;
        }

        /* define point array */
        Point host_sample[len];

        /* init point array */
        int i;
        for (i=0; i < len; i++) {
            /* get list the element -- float[] */
            jfloatArray element = (jfloatArray)(env->CallObjectMethod(objectList, list_get, i));
            if(element == NULL){
                cout << "fetch list element failure\n" << endl;
            }

            float *f_arrays;
            f_arrays = env->GetFloatArrayElements(element,NULL);
            if(f_arrays == NULL){
                cout << "fetch float array failure\n" << endl;
            }

            host_sample[i].id = i;
            host_sample[i].noise = -1;
            host_sample[i].cluster = -1;

            int j;
            int arr_len = static_cast<int>(env->GetArrayLength(element));
            for(j=0; j<arr_len ; j++){
                host_sample[i].dimensions[j] = f_arrays[j];
            }

            /* 释放可能复制的缓冲区 */
            env->ReleaseFloatArrayElements(element, f_arrays, 0);
            /* 调用 JNI 函数 DeleteLocalRef() 删除 Local reference。Local reference 表空间有限，这样可以避免 Local reference 表的内存溢出，避免 native memory 的 out of memory */
            env->DeleteLocalRef(element);
        }

        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, 0);

        /* core : run dbscan */
        host_algorithm_dbscan(host_sample, len, static_cast<float>(eps), static_cast<int>(minPts));

        hipEventRecord(end, 0);
        hipEventSynchronize(end);

        float time;
        hipEventElapsedTime(&time, start, end);
        cout<<"time: "<< time <<"ms --device\n"<<endl;

}